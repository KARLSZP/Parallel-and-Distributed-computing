#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "error_checks.h" // Macros CUDA_CHECK and CHECK_ERROR_MSG


__global__ void vector_add(double *C, const double *A, const double *B, int N)
{
    // Add the kernel code
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Do not try to access past the allocated memory
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}


int main(void)
{
    const int N = 20;
    const int ThreadsInBlock = 128;
    double *dA, *dB, *dC;
    double hA[N], hB[N], hC[N];
  
    for(int i = 0; i < N; ++i) {
        hA[i] = (double) i;
        hB[i] = (double) i * i;
    }

    /* 
       Add memory allocations and copies. Wrap your runtime function
       calls with CUDA_CHECK( ) macro
    */
    CUDA_CHECK( hipMalloc((void**)&dA, sizeof(double)*N) );
    // #error Add the remaining memory allocations and copies
    CUDA_CHECK( hipMalloc((void**)&dB, sizeof(double)*N) );
    CUDA_CHECK( hipMalloc((void**)&dC, sizeof(double)*N) );

    CUDA_CHECK( hipMemcpy((void*)dA, (void*)hA, sizeof(double)*N, hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy((void*)dB, (void*)hB, sizeof(double)*N, hipMemcpyHostToDevice) );
    
    // Note the maximum size of threads in a block
    dim3 threads(ThreadsInBlock), grid((N + threads.x - 1) / threads.x);

    //// Add the kernel call here
    // #error Add the CUDA kernel call
    // vector_add(double *C, const double *A, const double *B, int N);
    // // dereference host pointer hA, hB.
    // vector_add <<<grid, threads>>> (dC, hA, hB, N);
    
    vector_add <<<grid, threads>>> (dC, dA, dB, N);
    
    // Here we add an explicit synchronization so that we catch errors
    // as early as possible. Don't do this in production code!
    hipDeviceSynchronize();
    CHECK_ERROR_MSG("vector_add kernel");

    //// Copy back the results and free the device memory
    // #error Copy back the results and free the allocated memory
    CUDA_CHECK( hipMemcpy((void*)hC, (void*)dC, sizeof(double)*N, hipMemcpyDeviceToHost) );
    
    // // dereference device pointer dC[i]
    // for (int i = 0; i < N; i++)
    //     printf("%5.1f\n", dC[i]);

    for (int i = 0; i < N; i++)
        printf("%5.1f\n", hC[i]);

    return 0;
}